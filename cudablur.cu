#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

//Computes a single row of the destination image by summing radius pixels
//Parameters: src: Teh src image as width*height*bpp 1d array
//            dest: pre-allocated array of size width*height*bpp to receive summed row
//            row: The current row number
//            pWidth: The width of the image * the bpp (i.e. number of bytes in a row)
//            rad: the width of the blur
//            bpp: The bits per pixel in the src image
//Returns: None
__global__ void computeRow(uint8_t *dImg,uint8_t *mid,int pWidth,int height,int radius,int bpp){
	int i;
    	int bradius=radius*bpp;
	int row=blockIdx.x*blockDim.x+threadIdx.x;
	if(height>row){

    		//initialize the first bpp elements so that nothing fails
    		for (i=0;i<bpp;i++)
        		mid[row*pWidth+i]=dImg[row*pWidth+i];
    			//start the sum up to radius*2 by only adding (nothing to subtract yet)
    		for (i=bpp;i<bradius*2*bpp;i++)
        		mid[row*pWidth+i]=dImg[row*pWidth+i]+mid[row*pWidth+i-bpp];
     		for (i=bradius*2+bpp;i<pWidth;i++)
        		mid[row*pWidth+i]=dImg[row*pWidth+i]+mid[row*pWidth+i-bpp]-dImg[row*pWidth+i-2*bradius-bpp];
    			//now shift everything over by radius spaces and blank out the last radius items to account for sums at the end of the kernel, instead of the middle
    		for (i=bradius;i<pWidth;i++){
        		mid[row*pWidth+i-bradius]=mid[row*pWidth+i]/(radius*2+1);
    		}	
    		//now the first and last radius values make no sense, so blank them out
    		for (i=0;i<bradius;i++){
        		mid[row*pWidth+i]=0;
        		mid[(row+1)*pWidth-1-i]=0;
    		}
	}	

}

//Computes a single column of the destination image by summing radius pixels
//Parameters: src: Teh src image as width*height*bpp 1d array
//            dest: pre-allocated array of size width*height*bpp to receive summed row
//            col: The current column number
//            pWidth: The width of the image * the bpp (i.e. number of bytes in a row)
//            height: The height of the source image
//            radius: the width of the blur
//            bpp: The bits per pixel in the src image
//Returns: None
__global__ void computeColumn(uint8_t *dImg,uint8_t *mid,int pWidth,int height,int radius,int bpp){
	int i;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	if(pWidth>col){
		mid[col]=dImg[col];
    		//start tue sum up to radius*2 by only adding
    		for (i=1;i<=radius*2;i++)
        		mid[i*pWidth+col]=dImg[i*pWidth+col]+mid[(i-1)*pWidth+col];
    		for (i=radius*2+1;i<height;i++)
        		mid[i*pWidth+col]=dImg[i*pWidth+col]+mid[(i-1)*pWidth+col]-dImg[(i-2*radius-1)*pWidth+col];
    		//now shift everything up by radius spaces and blank out the last radius items to account for sums at the end of the kernel, instead of the middle
    		for (i=radius;i<height;i++){
        		mid[(i-radius)*pWidth+col]=mid[i*pWidth+col]/(radius*2+1);
    		}
   		 //now the first and last radius values make no sense, so blank them out
    		for (i=0;i<radius;i++){
        		mid[i*pWidth+col]=0;
        		mid[(height-1)*pWidth-i*pWidth+col]=0;
    		}
	}

}

//Usage: Prints the usage for this program
//Parameters: name: The name of the program
//Returns: Always returns -1
int Usage(char* name){
    printf("%s: <filename> <blur radius>\n\tblur radius=pixels to average on any side of the current pixel\n",name);
    return -1;
}

int main(int argc,char** argv){
    float t1,t2;
    int radius=0;
    int i;
    int width,height,bpp,pWidth;
    char* filename;
    uint8_t *img;
    uint8_t *dImg;
    float* dest,*mid,*hDest;

    if (argc!=3)
        return Usage(argv[0]);
    filename=argv[1];
    sscanf(argv[2],"%d",&radius);
   
    img=stbi_load(filename,&width,&height,&bpp,0);

    pWidth=width*bpp;  //actual width in bytes of an image row

    hipMalloc(&mid,sizeof(float)*pWidth*height);   
    cudeMalloc(&dest,sizeof(float)*pWidth*height);
    cudeMalloc(&dImg,sizeof(uint8_t)*pWidth*height);

    hipMemcpy(dImg,img,pWidth*height*sizeof(uint8_t),hipMemcpyHostToDevice);
    stbi_image_free(img);

    t1=clock();
    int blockSize=256;
    int gridSize=(pWidth+blockSize-1)/blockSize;
    computeColumn<<gridSize,blockSize>>(dImg,mid,pWidth,height,radius,bpp);

    hipDeviceSynchronize();
    blockSize=256;
    gridSize=(pWidth+blockSize-1)/blockSize;
    computeRow<<gridSize,blockSize>>(img,mid,pWidth,height,radius,bpp);
    hipDeviceSynchronize();
    hipFree(mid);
    t2=clock();

    hDest=(float*)malloc(sizeof(float)*pWidth*height);
    hipMemcpy(hDest,dest,sizeof(float)*pWidth*height,hipMemcpyHostToDevice);
    hipFree(dest);

    img=(uint8_t)malloc(sizeof(uint8_t)*pWidth*height);
    for(i=0;i<pWidth*height;i++){
	    img[i]=(uint8_t)hDest[i];
    }
    free(hDest);
    stbi_write_png("output.png",width,height,bpp,img,bpp*width);
    free(img);
    printf("Blur with radius %d complete in %f seconds\n",radius,(t2-t1)/CLOCKS_PER_SEC);
}
